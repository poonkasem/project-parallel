#include<iostream>
#include<stdio.h>
#include<string.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

using namespace std;
using namespace cv;

int main() {

	int i,j,k;

	cv::Mat input;
	input = cv::imread("input.jpg", IMREAD_COLOR);

	unsigned char *temp = (unsigned char*)(input.data);
	int rows = input.rows , cols = input.cols;
	int process[rows][cols];
	int swap[rows][cols];
	
	k=0;
	for(i=0;i<rows;i++){
		for(j=0;j<cols;j++){
			process[i][j] = temp[k];
			k++;
		}
	}

	int size_input = sizeof(char) * 3 * img_pic.rows * img_pic.cols;
	unsigned char *dev_input;
	hipMalloc( (void**)&dev_input, size_input);
	hipMemcpy( dev_input, temp, size_input, hipMemcpyHostToDevice);

	/*int colsTemp;
	for(i=0;i<rows;i++){
		k=0;
		colsTemp = cols-1;
		for(j=0;j<cols;j++){
			swap[i][k] = process[i][colsTemp];
			colsTemp--;
			k++;
		}
	}*/

	k=0;
	for(i=0;i<rows;i++){
		for(j=0;j<cols;j++){
			temp[k] = swap[i][j];
			k++;
		}
	}
	
	Mat output =  Mat(rows, cols, CV_8UC3 , temp);
	cv::imwrite("output.jpg",output);
}