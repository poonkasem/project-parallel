#include "hip/hip_runtime.h"
#include<stdio.h>
#include<opencv2/core/core.hpp>
#include<opencv2/highgui/highgui.hpp>
#include<iostream>

#define T 16
#define VISIBILITY 70 // watermark transparency level

using namespace cv;
using namespace std;

__global__ void addwatermark(unsigned char *pic, unsigned char *mark, int h_pic, int w_pic, int h_mark, int w_mark){
	//as each global threadid number
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	//finding each thread pixel to calculation
	int in_m = y * w_mark + x;
	//prevent pixel of thread that out of mark boundary to calculate
	int in_p = (h_pic - h_mark + y) * w_pic + x;

	//if that pixel is in watermark boundary
	if((x<w_mark)&&(y<h_mark))
	{
		if(mark[in_m*3]==0 && mark[in_m*3+1]==0 && mark[in_m*3+2]==0)
		{
			//do nothing png will be transparent
		}
		else
		{	//if there is color inside finding each B,G,R and blend it with picture that we want
			//*0.01 to convert 100% that multiply back to normal range
			pic[in_p*3] = ((pic[in_p*3]*VISIBILITY)+(mark[in_m*3]*(100-VISIBILITY)))*0.01;
			pic[in_p*3+1] = ((pic[in_p*3+1]*VISIBILITY)+(mark[in_m*3+1]*(100-VISIBILITY)))*0.01;
			pic[in_p*3+2] = ((pic[in_p*3+2]*VISIBILITY)+(mark[in_m*3+2]*(100-VISIBILITY)))*0.01;
		}
	}
}

int main(int argc, char* argv[]){
	// load both image and watermark
	Mat img_pic = imread(argv[1], IMREAD_COLOR);
	Mat img_mark = imread(argv[2], IMREAD_COLOR);

	//checking for prevent that watermrk will be bigger and can't fit in
	if(img_pic.rows < img_mark.rows || img_pic.cols < img_mark.cols )
	{
		cout <<  "Size of watermark is bigger than wallpaper" << endl;
        	return -1;
	}

	// show Original Image
	//imshow("Original", img_pic);

	// convert datatype of image from Mat to unsigned char
	unsigned char *in_pic = (unsigned char*)(img_pic.data);
	unsigned char *in_mark = (unsigned char*)(img_mark.data);

	// allocate global memory space in GPU for using according to rows and column
	int size_pic = sizeof(char) * 3 * img_pic.rows * img_pic.cols;
	int size_mark = sizeof(char) * 3 * img_mark.rows * img_mark.cols;

	unsigned char *dev_pic, *dev_mark;
	
	hipMalloc( (void**)&dev_pic, size_pic);
	hipMalloc( (void**)&dev_mark, size_mark);

	// copy data from cpu to gpu
	hipMemcpy( dev_pic, in_pic, size_pic, hipMemcpyHostToDevice);
	hipMemcpy( dev_mark, in_mark, size_mark, hipMemcpyHostToDevice);

	// set number of thread and block to use
	dim3 dimblock(T, T);
	dim3 dimgrid((img_mark.cols + dimblock.x - 1)/dimblock.x, (img_mark.rows + dimblock.y - 1)/dimblock.y);

	// call kernel routine send row column and pic
	addwatermark<<<dimgrid, dimblock>>>(dev_pic, dev_mark, img_pic.rows, img_pic.cols, img_mark.rows, img_mark.cols);

	// copy back after calculation from GPU to CPU
	hipMemcpy( in_pic, dev_pic, size_pic, hipMemcpyDeviceToHost);

	// convert datatype back to print out unsigned char to Mat
	Mat out =  Mat(img_pic.rows, img_pic.cols, CV_8UC3, in_pic);

	// free memory space in GPU
	hipFree(dev_pic);
	hipFree(dev_mark);

	// write image
	imwrite("output.jpg", out);
	//imshow("Modified", img_output);

	//displays the image for specified milliseconds
	waitKey();

	return 0;
}