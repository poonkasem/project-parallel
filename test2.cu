#include<iostream>
#include<stdio.h>
#include<string.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/core/mat.hpp>

using namespace std;
using namespace cv;

int main() {

	int i,j;

	cv::Mat input;
	input = cv::imread("input.jpg", IMREAD_COLOR);

    int rows = input.rows, cols = input.cols;
    
    cv::Mat output;

    output = input;

    unsigned char *in = (unsigned char*)(input.data);
    unsigned char *out = (unsigned char*)(output.data);

    int sizeUnsignedChar = sizeof(char)*3*rows*cols;

    unsigned char *devInput, *devOutput;

    hipMalloc((void**)&devInput, sizeUnsignedChar);
    hipMalloc((void**)&devOutput, sizeUnsignedChar);

    hipMemcpy( devInput, in, sizeUnsignedChar, hipMemcpyHostToDevice);
    hipMemcpy( devOutput, out, sizeUnsignedChar, hipMemcpyHostToDevice);

    /*for (i=0;i<rows;i++) {
        for (j=0;j<cols;j++) {
            // output(i,j) = input(i,cols-1-j);
            out[]
        }
    }

	cv::imwrite("output.jpg",output);*//

}